#include "hip/hip_runtime.h"
__global__ void suma_vectores(const float *a, const float *b,   float*resultado, int tamanio){

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index<tamanio){
        resultado[index] = a[index]+b[index];
    }

}

extern "C" void suma(const float *a, const float *b,   float*resultado, int tamanio) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kerbel_bordes_image<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}

