

#include <hip/hip_runtime.h>
#include <iostream>


__global__ void kernel_neagtivo(unsigned char* image, unsigned char* dst_image, int width, int height ){

    int index = blockDim.x*blockIdx.x + threadIdx.x;
    int pix_y = index / width;
    int pix_x = index % width;

    int id = (pix_y*width+pix_x)*4;

    dst_image[id] = 255- image[id];
    dst_image[id+1] = 255- image[id+1];
    dst_image[id+2] = 255- image[id+2];
    dst_image[id+3] = 255;

}
extern "C" void kernel_negativo_ex(unsigned char* src_image, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kernel_neagtivo<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}