
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void kernel_min(unsigned char* image, unsigned char* image2, int width, int height ){

    unsigned char r[9];
    unsigned char g[9];
    unsigned char b[9];
    int contador=0;

    int index = blockDim.x*blockIdx.x + threadIdx.x;
    int pix_y = index / width;
    int pix_x = index % width;

    for(int i=pix_x-1;i<=pix_x+1;i++) {
        for(int j=pix_y-1;j<=pix_y+1;j++) {
            int id = (j * width + i) * 4;

            if (i >= 0 && i < width && j >= 0 && j < height) {
                r[contador] = (image[id]);
                g[contador] = (image[id + 1]);
                b[contador] = (image[id + 2]);

                contador++;
            }
        }

    }
    unsigned int minR= r[0];
    unsigned int minG=  g[0];
    unsigned int minB = b[0];

    for(int i = 0; i < contador; i++  ){
        minR = min(minR, r[i]);
        minG = min(minG, g[i]);
        minB = min(minB, b[i]);
    }
    int index_final = (pix_y * width + pix_x) * 4;

    image2[index_final] = minR;
    image2[index_final + 1] = minG;
    image2[index_final + 2] = minB;
    image2[index_final + 3] = 255;


}
extern "C" void kernel_min_ex(unsigned char* src_image, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kernel_min<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}