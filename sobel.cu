
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void kernel_sobel(unsigned char* image, unsigned char* dst_image, int width, int height ){

    int index = blockDim.x*blockIdx.x + threadIdx.x;
    int pix_y = index / width;
    int pix_x = index % width;

    int r_x = 0;
    int g_x = 0;
    int b_x = 0;

    int r_y = 0;
    int g_y = 0;
    int b_y = 0;

    int sobel_x[] = { -1, 0, 1, -2, 0, 2, -1, 0, 1 };
    int sobel_y[] = { -1, -2, -1, 0, 0, 0, 1, 2, 1 };

    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int index = ((pix_y* width + pix_x) * 4) + (i * 4) + (j * width * 4);

            if (index >= 0 && index <= width * height * 4) {
                int matrixIndex = (i + 1) * 3 + (j + 1);

                int weight_x = sobel_x[matrixIndex];
                int weight_y = sobel_y[matrixIndex];

                r_x += image[index] * weight_x;
                g_x += image[index + 1] * weight_x;
                b_x += image[index + 2] * weight_x;

                r_y += image[index] * weight_y;
                g_y += image[index + 1] * weight_y;
                b_y += image[index + 2] * weight_y;
            }
        }
    }

    int r = min(max(abs(r_x)+ abs(r_y)/2, 0), 255);
    int g =  min(max(abs(g_x)+ abs(g_y)/2, 0), 255);
    int b =  min(max(abs(b_x)+ abs(b_y)/2, 0), 255);

    int posicion = (pix_y*width+pix_x)*4;


    dst_image[posicion] = r;
    dst_image[posicion + 1] = g;
    dst_image[posicion + 2] = b;
    dst_image[posicion + 3] = 255;




}
extern "C" void kernel_sobel_ex(unsigned char* src_image, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kernel_sobel<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}