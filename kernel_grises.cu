

#include <hip/hip_runtime.h>
#include <iostream>


__global__ void kernel_grises(unsigned char* image, unsigned char* dst_image, int width, int height ){

    int index = blockDim.x*blockIdx.x + threadIdx.x;
    int pix_y = index / width;
    int pix_x = index % width;

    int id = (pix_y * width + pix_x) * 4;
    int escala_grises = image[id] * 0.299 + image[id + 1] * 0.587 + image[id + 2] * 0.114;
    dst_image[id] = escala_grises;
    dst_image[id + 1] = escala_grises;
    dst_image[id + 2] = escala_grises;
    dst_image[id + 3] = 255;

}
extern "C" void kernel_grises_ex(unsigned char* src_image, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kernel_grises<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}