
#include <hip/hip_runtime.h>
#include <stdio.h>

#define image_channels 4


__global__ void kerbel_bordes_image(unsigned char* src_image, unsigned char* dst_image, int width, int height) {

    int index = blockDim.x*blockIdx.x + threadIdx.x;

    int pix_y = index / width;
    int pix_x = index % width;


    int r = 0;
    int g = 0;
    int b = 0;
    int cc = 0;

    int matriz[] = { 0, 1, 0, 1, -4, 1, 0, 1, 0 };

    for(int i=-1;i<=1;i++) {
        for(int j=-1;j<=1;j++) {
            int index = (pix_y * width + pix_x)*image_channels + (i * 4) + (j * width * 4);

            if(index >= 0 && index <= width * height * image_channels) {
                int matrixIndex = (i + 1) * 3 + (j + 1);
                int weight = matriz[matrixIndex];

                r += src_image[index] * weight;
                g += src_image[index + 1] * weight;
                b += src_image[index + 2] * weight;
            }
        }
    }

    int index_final = (pix_y * width + pix_x)*image_channels;
    r = max(0,min(r,255));
    g = max(0,min(g,255));
    b = max(0,min(b,255));

    dst_image[index_final+0] = r;
    dst_image[index_final+1] = g;
    dst_image[index_final+2] = b;
    dst_image[index_final+3] = 255;
}

extern "C" void kernel_bordes(unsigned char* src_image, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kerbel_bordes_image<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}