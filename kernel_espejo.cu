
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void kernel_espejo(unsigned char* image, unsigned char* dst_image, int width, int height ){

    int index = blockDim.x*blockIdx.x + threadIdx.x;
    int pix_y = index / width;
    int pix_x = index % width;

    int start =(pix_y*width*4);
    int end = start+(width-1)*4;
    int condicion = end;
    while (start < condicion) {
        dst_image[start] = image[end - 4];
        dst_image[start + 1] = image[end - 3];
        dst_image[start + 2] = image[end - 2];
        dst_image[start + 3] = image[end-1];
        start += 4;
        end -= 4;
    }
}
extern "C" void kernel_espejo_ex(unsigned char* src_image, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kernel_espejo<<<blk_in_grid,thr_per_blk>>>(src_image, dst_image, width, height);
}