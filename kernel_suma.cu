
#include <hip/hip_runtime.h>
#include <iostream>

/*
 std::vector<sf::Uint8> sumaImagen_Serial(const sf::Uint8* imagen, const sf::Uint8* imagen2, int with, int height){

    std::vector<sf::Uint8> buffer(with*height*4);
    for(int i= 0; i<height; i++){
        for(int j =0; j<with; j++){

            int index= (i*with+j)*image_channels;
            buffer[index] = (imagen[index]+ imagen2[index])/2;
            buffer[index+1] = (imagen[index+1]+ imagen2[index+1])/2;
            buffer[index+2] = (imagen[index+2]+ imagen2[index+2])/2;;
            buffer[index+3] = 255;


        }

    }
    return buffer;
}
 */

__global__ void kernel_sum(unsigned char* image,unsigned char* image2, unsigned char*
dst_image, int width, int height){

    int index = blockDim.x*blockIdx.x + threadIdx.x;


    int y = index/width;
    int x = index % width;

    int posicion = (y*width+x)*4;

    dst_image[posicion] = (image[posicion]+ image2[posicion])/2;
    dst_image[posicion+1] = (image[posicion+1]+ image2[posicion+1])/2;
    dst_image[posicion+2] = (image[posicion+2]+ image2[posicion+2])/2;;
    dst_image[posicion+3] = 255;



}
extern "C" void kernel_sum_ex(unsigned char* src_image, unsigned char* src_image2, unsigned char* dst_image, int width, int height) {
    //kernel
    int thr_per_blk = 1024;//256;
    int blk_in_grid = ceil( float(width*height) / thr_per_blk );

    kernel_sum<<<blk_in_grid,thr_per_blk>>>(src_image, src_image2, dst_image, width, height);
}